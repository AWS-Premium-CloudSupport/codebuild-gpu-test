
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello_kernel() {
    printf("Hello from GPU!\n");
}

int main() {
    hello_kernel<<<1,1>>>();
    hipDeviceSynchronize();
    return 0;
}
